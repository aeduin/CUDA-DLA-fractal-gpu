
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <fstream>
#define CIRCLE_BORDER -1
//1000 * 1

#define RANDOM_WALK

typedef struct {
    float x;
    float y;
    float horizontal_speed;
    float vertical_speed;
    uint seed;
} Particle;

typedef struct {
    int x;
    int y;
} Veci2D;

const float radius = 2.0f;
const int ceil_radius = (int)radius + ((((float)(int)radius) < radius) ? 1 : 0);
const float max_speed = 3.0f;
const int particle_count = 4096 * 8;

const int grid_size = 1024 * 2;
const int grid_width = grid_size;
const int grid_height = grid_size;

using ullong = unsigned long long;

__device__ int grid[grid_height][grid_width];
__constant__ Veci2D* circle_indices;
__constant__ int circle_indices_length;
__device__ int border_left;
__device__ int border_right;
__device__ int border_top;
__device__ int border_bottom;
__device__ int smallest_distance_to_center;

__device__ ullong total_static_particles;
__device__ ullong weight_center_x;
__device__ ullong weight_center_y;

__device__ int debug = 0;
__device__ int debug_array[1024];

void VecAdd();
void simulate();
void tick(Particle* particles, int tick_count);
__host__ __device__ int random_int(int min, int max, uint seed);
__device__ Particle make_static(Particle particle, int tick_count, float modulo_x, float modulo_y);
__host__ __device__ float random_float(uint seed);

#define print(message) std::cout << message << std::endl

int main() {
    print("starting");
    simulate();
    print("done");
}

// checks for cuda errors
// could be improved
void cuda_error() {
    auto result = hipGetLastError();
    if (result != hipSuccess) {
        do {
            std::cout << "error: " << result << std::endl;
            std::cout << "error message: " << hipGetErrorString(result) << std::endl;

            result = hipGetLastError();
            break;
        }
        while(result != hipSuccess);
    }
    else {
        std::cout << "success" << std::endl;
    }
}

// sets the grid values to -1
__global__ void init_grid_negative() {
    grid[blockIdx.y * blockDim.y + threadIdx.y][blockIdx.x * blockDim.x + threadIdx.x] = -1;
}

// sets the center of the grid to 0
__global__ void init_grid_center() {
    border_top = grid_height / 2;
    border_bottom = grid_height / 2;
    border_left = grid_width / 2;
    border_right = grid_width / 2;
    smallest_distance_to_center = CIRCLE_BORDER * CIRCLE_BORDER;

    if(CIRCLE_BORDER < 0) {
        grid[grid_height / 2][grid_width / 2] = 0;
    }
    else {
        // init weight center
        int center_bias = 10;
        total_static_particles = center_bias;
        weight_center_x = (grid_width / 2) * center_bias;
        weight_center_y = (grid_height / 2) * center_bias;
    }
}

// outputs the grid (and its widht/height) to a file
void output_grid() {
    // get grid from GPU memory
    size_t mem_size = sizeof(int) * grid_height * grid_width;
    int* host_grid = (int*)malloc(mem_size);
    hipMemcpyFromSymbol(host_grid, HIP_SYMBOL(grid), mem_size, 0, hipMemcpyDeviceToHost);

    // create file
    std::ofstream output_file;
    output_file.open("grid_output.bin", std::ios::binary);
    if(output_file.is_open()) {
        print("output_file is open");
    }

    // output to file
    const int ints[2] = {grid_width, grid_height};
    output_file.write((const char*) &ints, sizeof(int) * 2);
    //output_file.write((const char*) &grid_height, sizeof(int));
    output_file.write((const char*) host_grid, mem_size);


    // std::cout << std::endl << std::endl << "[";
    // for(int y = 0; y < grid_height; y++) {
    //     std::cout << "\"";
    //     for(int x = 0; x < grid_width; x++) {
    //         int value_at_xy = *(host_grid + x + y * grid_width);
    //         // std::cout << value_at_xy << ",";
    //         //std::cout << (value_at_xy >= 0) ? "1" : "0";
    //         print(value_at_xy);
    //     } 
    //     std::cout << "\"," << std::endl;
    // }
    // std::cout << "]" << std::endl << std::endl;
    
    // clean up
    output_file.close();
    delete host_grid;
}

__host__ __device__ uint hash(uint x) {
    const uint seed = 1324567967;
    x += seed;
    x = ((x >> 16) ^ x) * seed;
    x = ((x >> 16) ^ x) * seed;
    x = (x >> 16) ^ x;
    return x;
}

// returns an int in the range [min, max) based on seed
__host__ __device__ int random_int(int min, int max, uint seed) {
    uint random = hash(seed);
    random %= (uint)(max - min);
    
    return (int)random + min;
}

// returns a float in the range [0, 1) based on seed;
__host__ __device__ float random_float(uint seed) {
    const int max = 10000000;
    int base = random_int(0, max, seed);

    return fmodf((float)base / (float)max, 1.0);
}

__device__ Particle randomize_speed(Particle particle, int direction_seed, int speed_seed) {
    float direction = M_PI * 2.0f * random_float(direction_seed);
    float speed = random_float(speed_seed) * max_speed;

    particle.vertical_speed = cosf(direction) * speed;
    particle.horizontal_speed = sinf(direction) * speed;

    return particle;
}

// randomizes all fields of the particle
__device__ Particle randomize_particle(Particle particle) {
    uint seed = particle.seed;
    int center_width = border_right - border_left;
    int center_height = border_bottom - border_top;

    if(CIRCLE_BORDER < 0) {
        particle.x = random_int(0, grid_width, seed + 0);

        if(particle.x > border_left && particle.x < border_right) {
            particle.y = random_int(0, grid_height - center_height, seed + 1);

            if(particle.y > border_top) {
                particle.y += center_height;
            }
        }
        else {
            particle.y = random_int(0, grid_height, seed + 1);
        }
    }
    else {
        particle.x = (float) (grid_width - (weight_center_x / total_static_particles));
        particle.y = (float) (grid_height - (weight_center_y / total_static_particles));
        // particle.x = grid_width / 2;
        // particle.y = grid_height / 2;
        // debug = (int) particle.x;
    }

    particle = randomize_speed(particle, seed + 2, seed + 3);

    particle.seed = hash(seed);

    return particle;
}

// initializes the particle
__global__ void init_particles(Particle* particles) {
    int i = threadIdx.x + blockIdx.x * blockDim.x; // particle index in the particles array
    Particle* particle = particles + i;
    particle->seed = (uint)i * 4;
    *particle = randomize_particle(*particle);
}

// prints border_left, border_right, border_top and border_bottom to stdio
void print_boundaries() {
    int left, right, top, bottom;

    hipMemcpyFromSymbol(&left, HIP_SYMBOL(border_left), sizeof(int));
    hipMemcpyFromSymbol(&right, HIP_SYMBOL(border_right), sizeof(int));
    hipMemcpyFromSymbol(&top, HIP_SYMBOL(border_top), sizeof(int));
    hipMemcpyFromSymbol(&bottom, HIP_SYMBOL(border_bottom), sizeof(int));
    print(left << ", " << right << ", " << top << ", " << bottom);
}

void simulate() {
    // initialize grid

    cuda_error();

    dim3 threadsPerBlock(16, 16);
    dim3 blocks(grid_width / threadsPerBlock.x, grid_height / threadsPerBlock.y);
    init_grid_negative<<<blocks, threadsPerBlock>>>();
    init_grid_center<<<1, 1>>>();

    // initialize particles
    size_t mem_size = particle_count * sizeof(Particle);
    Particle* particles;
    hipMalloc(&particles, mem_size);
    const int particle_threads_per_block = 256;
    const int particle_blocks = particle_count / particle_threads_per_block;

    cuda_error();
    init_particles<<<particle_blocks, particle_threads_per_block>>>(particles);
    // done intializing particles

    print_boundaries();
    cuda_error();

    int tick_count = 0;
    for(int i = 0; true; i++) {
        tick(particles, ++tick_count);

      
        int left, right, top, bottom, center_distance;
        int debug_copy;
        int debug_array_copy[1024];
        ullong total_static_particles_copy;

        hipMemcpyFromSymbol(&left, HIP_SYMBOL(border_left), sizeof(int));
        hipMemcpyFromSymbol(&right, HIP_SYMBOL(border_right), sizeof(int));
        hipMemcpyFromSymbol(&top, HIP_SYMBOL(border_top), sizeof(int));
        hipMemcpyFromSymbol(&bottom, HIP_SYMBOL(border_bottom), sizeof(int));
        hipMemcpyFromSymbol(&bottom, HIP_SYMBOL(border_bottom), sizeof(int));
        hipMemcpyFromSymbol(&center_distance, HIP_SYMBOL(smallest_distance_to_center), sizeof(int));
        hipMemcpyFromSymbol(&debug_copy, HIP_SYMBOL(debug), sizeof(int));
        hipMemcpyFromSymbol(&total_static_particles_copy, HIP_SYMBOL(total_static_particles), sizeof(ullong));

        hipMemcpyFromSymbol(&debug_array_copy, HIP_SYMBOL(debug_array), sizeof(int) * debug_copy);

        if(i % 10000 == 0) {
            print(left << ", " << right << ", " << top << ", " << bottom << ", " << center_distance);
            print(debug_copy);
            // print(total_static_particles_copy);

            for(int i = 0; i < debug_copy && i < 1024; i++) {
                if(i % 2 == 0) {
                    print("");
                }
                print(debug_array_copy[i]);
            }
        }
        const int margin = 100;
        if(CIRCLE_BORDER > -1 && center_distance < margin * margin) {
            break;
        }
        if(left < margin || right > grid_width - margin || top < margin || bottom > grid_height - margin) {
            break;
        }
    }
    cuda_error();
    output_grid();

    hipFree(particles);
}

__device__ float pythagoras(float a, float b) {
    return a * a + b * b;
}

__device__ float pythagoras(Particle particle) {
    return pythagoras(particle.x - (float)(grid_width / 2), particle.y - (float)(grid_height / 2));
}

__device__ Particle move_particle(Particle particle) {
    
    #ifdef RANDOM_WALK
    // randomize direction and speed
    particle = randomize_speed(particle, particle.seed, particle.seed + 1);
    particle.seed = hash(particle.seed);
    #endif

    // move particle
    particle.x += particle.horizontal_speed;
    particle.y += particle.vertical_speed;

    // check bounds
    if(particle.x - radius <= 0.0f) {
        particle.x = 0.01f + radius;
        particle.horizontal_speed *= -1.0f;
    }
    else if(particle.x + radius >= grid_width) {
        particle.x = grid_width - 0.01f - radius;
        particle.horizontal_speed *= -1.0f;
    }
    if(particle.y - radius <= 0.0f) {
        particle.y = 0.01f + radius ;
        particle.vertical_speed *= -1.0f;
    }
    else if(particle.y + radius >= grid_height) {
        particle.y = grid_height - 0.01f - radius;
        particle.vertical_speed *= -1.0f;
    }

    return particle;
}

__global__ void particle_step(Particle* particles, int tick_count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // particle index in the particles array
    Particle particle = particles[i];

    // calculate some variable values to be used later
    const int diameter = ceil_radius * 2;

    const int max_steps = 1;
    // move at least once
    bool outside_border_margins = true;
    const int border_margins = 250;

    if(CIRCLE_BORDER < 0) {
        for(int i = 0; i < max_steps && outside_border_margins; i++) {
            particle = move_particle(particle);
            outside_border_margins = particle.x < (border_left - border_margins) || particle.x > (border_right + border_margins) || particle.y > (border_bottom + border_margins) || particle.y < (border_top - border_margins);
        }
    }
    else {
        // set to false to avoid confusion
        outside_border_margins = false;
        particle = move_particle(particle);
    }

    float modulo_x = fmod(particle.x, 1.0f);
    float modulo_y = fmod(particle.y, 1.0f);

    if(CIRCLE_BORDER > -1 && (int)(pythagoras(particle) + radius) >= CIRCLE_BORDER * CIRCLE_BORDER) {
        particles[i] = make_static(particle, tick_count, modulo_x, modulo_y);
        return;
    }

    // if(true) {
    if(!outside_border_margins) {
        bool looping = true;
        
        for(int dx = -ceil_radius; dx <= ceil_radius && looping; dx++) {
            for(int dy = -ceil_radius; dy <= ceil_radius && looping; dy++) {
                // calculate distance from center of the particle
                float distance_x = -dx + modulo_x;
                float distance_y = -dy + modulo_y;
    
                // if(pythagoras(distance_x, distance_y) < radius * radius && pythagoras(abs(distance_x) + 1, abs(distance_y) + 1) > radius * radius) {

                if(pythagoras(distance_x, distance_y) < radius * radius) {
                    // position is within distance of the center
                    if(grid[(int)(particle.y - distance_y)][(int)(particle.x - distance_x)] >= 0) {
                        // it hit another particle
                        particle = make_static(particle, tick_count, modulo_x, modulo_y);
    
                        looping = false;
                        break;
                    }
                }
            }
        }
    }

    particles[i] = particle;
}

__device__ Particle make_static(Particle particle, int tick_count, float modulo_x, float modulo_y) {
    for(int dx2 = -ceil_radius; dx2 <= ceil_radius; dx2++) {
        for(int dy2 = -ceil_radius; dy2 <= ceil_radius; dy2++) {
            // calculate distance from center of the particle
            float distance_x2 = -dx2 + modulo_x;
            float distance_y2 = -dy2 + modulo_y;

            if(distance_x2 * distance_x2 + distance_y2 * distance_y2 < radius * radius) {
                // calculate position in grid
                int absolute_x = (int)(particle.x - distance_x2);
                int absolute_y = (int)(particle.y - distance_y2);
                
                // if the absolute_x/y are within the grid
                if(absolute_x >= 0 && absolute_x < grid_width && absolute_y >= 0 && absolute_y < grid_height) {
                    // set the grid to being hit
                    grid[absolute_y][absolute_x] = tick_count;

                    /*
                        Because the program writes and reads from the same grid in a single tick,
                        the algorithm isn't completely deterministic. I could use two different 
                        grids and then copy values, but it doesn't feel necessary.
                    */
                }
            }
        }
    }

    if(CIRCLE_BORDER < 0) {
        atomicMin(&border_left, (int)(particle.x - radius));
        atomicMax(&border_right, (int)(particle.x + radius));
        atomicMin(&border_top, (int)(particle.y - radius));
        atomicMax(&border_bottom, (int)(particle.y + radius));
    }
    else {
        atomicMin(&smallest_distance_to_center, (int)(pythagoras(particle) - radius));
        atomicAdd(&total_static_particles, 1l);
        atomicAdd(&weight_center_x, (ullong)particle.x);
        atomicAdd(&weight_center_y, (ullong)particle.y);
    }

    // give the particle a random new position and speed
    return randomize_particle(particle);
}

// perform one tick
void tick(Particle* particles, int tick_count) {
    const int threads_per_block = 16;
    const int blocks = particle_count / threads_per_block;

    particle_step<<<blocks, threads_per_block>>>(particles, tick_count);
}
